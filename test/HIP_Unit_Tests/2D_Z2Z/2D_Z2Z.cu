#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 256 
#define NY 128  
#define BATCH 1

int main()
{
	hipfftHandle plan;
	int Csize = ((NX/2) + 1) * NY;
	hipfftDoubleComplex *input = (hipfftDoubleComplex*)calloc(Csize, sizeof(hipfftDoubleComplex));
	hipfftDoubleComplex *output = (hipfftDoubleComplex*)calloc(Csize, sizeof(hipfftDoubleComplex));
	hipfftDoubleComplex *idata;
	hipfftDoubleComplex *odata;

	int seed = 123456789;
	srand(seed);

	for(int i=0; i<Csize; i++)
	{
		input[i].x = rand();
		input[i].y = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftDoubleComplex)*Csize*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftDoubleComplex)*Csize*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftDoubleComplex)*Csize*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftDoubleComplex)*Csize*BATCH, hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to allocate\n"); 
		return 0;
	} 

	/* Create a 2D FFT plan. */ 
	if (hipfftPlan2d(&plan, NX, NY, HIPFFT_Z2Z) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;
	}

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)idata, (hipfftDoubleComplex*)odata, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		return 0;	
	}

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)idata, (hipfftDoubleComplex*)odata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
		return 0;	
	}

	if (hipDeviceSynchronize() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
		return 0; 
	}

	hipfftDestroy(plan); 
	
	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata);

	return 0;
}

